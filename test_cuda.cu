
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

#define N 10

using namespace std;

__global__
void saxpy_cuda (int n, float a, float * x, float * y) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    /*if (i < n)*/ y[i] = a * x[i] + y[i];
}

int main () {

    int size = N * sizeof(float);

    int active_gpu_id;
    int gpus_count;
    hipDeviceProp_t gpu_props;

    hipGetDeviceCount(&gpus_count);

    if (gpus_count) {

        hipGetDevice(&active_gpu_id);

        // char* name
        // int maxThreadsPerBlock;
        // int maxThreadsDim[3];
        // int maxGridSize[3];
        // int integrated
        // int pciBusID
        // int pciDeviceID

        cout << "There is " << gpus_count << " GPUs available on your machine which are :" << endl;

        for (int i = 0; i < gpus_count; i++) {

            hipGetDeviceProperties(&gpu_props, i);
            cout << "- " << gpu_props.name << " (id=" << i << ")";
            if (gpu_props.integrated) cout << " [INTEGRATED]";
            if (i == active_gpu_id) cout << " [ACTIVE]";
            cout << endl;
            cout << "---> maxThreadsPerBlock = " << gpu_props.maxThreadsPerBlock << endl;
            cout << "---> maxThreadsDim = (" << gpu_props.maxThreadsDim[0] << ", " << gpu_props.maxThreadsDim[1] << ", " << gpu_props.maxThreadsDim[2] << ")" << endl;
            cout << "---> maxGridSize = (" << gpu_props.maxGridSize[0] << ", " << gpu_props.maxGridSize[1] << ", " << gpu_props.maxGridSize[2] << ")" << endl;
        }

        // cout << endl;
        // cout << "Wanna change GPU?" << endl;

        // cudaSetDevice(int device);

        // Host
        float x_h[N];
        float y_h[N];

        for (int i = 0; i < N; i++) {

            x_h[i] = i;
            y_h[i] = i + N;
        }

        // Device
        float * x_d;
        float * y_d;

        // (void **)
        hipMalloc(&x_d, size);
        hipMalloc(&y_d, size);

        hipMemcpy(x_d, x_h, size, hipMemcpyHostToDevice);
        hipMemcpy(y_d, y_h, size, hipMemcpyHostToDevice);

        saxpy_cuda <<<N, 1>>> (N, 2.0, x_d, y_d);

        hipMemcpy(y_h, y_d, size, hipMemcpyDeviceToHost);

        hipFree(x_d);
        hipFree(y_d);

        cout << "RESULT :";

        for (int i = 0; i < N; i++)
            cout << " " << y_h[i];

        cout << endl;
    }

    else
        cout << "Sorry but no GPU available on your machine" << endl;

    return EXIT_SUCCESS;
}