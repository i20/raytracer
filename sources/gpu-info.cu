
#include <hip/hip_runtime.h>
#include <cstdlib>

#include <iostream>

using namespace std;

int main () {

    int active_gpu_id;
    int gpus_count;
    hipDeviceProp_t gpu_props;

    hipGetDeviceCount(&gpus_count);

    if (gpus_count) {

        hipGetDevice(&active_gpu_id);

        cout << "There is " << gpus_count << " GPUs available on your machine which are :" << endl;

        for (int i = 0; i < gpus_count; i++) {

            hipGetDeviceProperties(&gpu_props, i);
            cout << "- " << gpu_props.name << " (id=" << i << ")";
            if (gpu_props.integrated) cout << " [INTEGRATED]";
            if (i == active_gpu_id) cout << " [ACTIVE]";
            cout << endl;
            cout << "---> maxThreadsPerBlock = " << gpu_props.maxThreadsPerBlock << endl;
            cout << "---> maxThreadsDim = (" << gpu_props.maxThreadsDim[0] << ", " << gpu_props.maxThreadsDim[1] << ", " << gpu_props.maxThreadsDim[2] << ")" << endl;
            cout << "---> maxGridSize = (" << gpu_props.maxGridSize[0] << ", " << gpu_props.maxGridSize[1] << ", " << gpu_props.maxGridSize[2] << ")" << endl;
        }
    }

    else
        cout << "Sorry but no GPU available on your machine" << endl;

    return EXIT_SUCCESS;
}
